#include "hip/hip_runtime.h"
#include "file.h"

extern __device__ __managed__ int PAGEFAULT ; 
extern __device__ __managed__ u32 inTime ; 
extern __device__ __managed__ uchar storage[] ; 

__device__ u32 paging( uchar *buffer, u32 frame_num, u32 offset ){
	u32 target ; 
	int pt_entries = PT_ENTRIES ;
	/* 
		The format of entry :
		1. Bit 0 is used to store valid/invalid bit 
		2. From bit 1 to 12 is used to store logical page number
		3. From bit 13 to 31 is used to store clock time
	*/
	/* 這裡是用來找有沒有重複 hit 的page*/ 
	for(int i = 0; i < pt_entries; ++i ){
		u32 mask = ( (1<<13) - 2 ) ; 
		u32 pageNum = ( pt[i] & mask ) >> 1 ; 
		/* pageNum 用來存pt[i]的logic page number */
		
		/* If frame_num(the logic page number want to query) 
			is the same as logical page number in entry   
		*/
		if( ( pt[i] & 1 )  && pageNum == frame_num ){
			u32 tmpTime = inTime++ ;
			// update hit time  	
			pt[i] = ( tmpTime << 13 ) | ( frame_num << 1 ) | 1 ; 
			return i * 32 + offset ;
		}
	}
	
	for(int i = 0; i < pt_entries; ++i ){
		if( (~pt[i]) & 1 )	{	// If find invalid entry( empty entry )
			PAGEFAULT++ ;	// add PageFault
			/*
				update page table
			*/
			u32 tmpTime = inTime++ ;
			pt[i] = ( tmpTime << 13 ) | ( frame_num << 1 ) | 1 ; 
			return i * 32 + offset  ; 
		}
	}

	u32 timeRange = 0 ; 
	// timeRange = CurrentTime - hitPageTime
	// timeRange is used to determine what the least time is 
	// if some time is earlier, the timeRange is wider
	// target variable is used to store the entry
	for(int i = 0; i < pt_entries; ++i ){
		u32 mask = (u32)(-1) ; 
		u32 tmpTime  = ( mask & pt[i] ) >> 13 ;
		u32 tmpTimeRange = inTime - tmpTime ; 

		if( tmpTimeRange > timeRange  ){
			target = i ;
			timeRange = tmpTimeRange ;  
		}
	}
	
	PAGEFAULT++ ;
	/*
		move the page from shared memory to global memory 
		And move the page form secondary storage to shared memory 
	*/
	u32 mask = ( 1 << 13 ) - 2 ; 
	u32 tarFrame = ( pt[target] & mask) >> 1 ;	//要被換掉的logical page
	u32 beginAddress = tarFrame * 32; //要被換掉的page的目標secondary memory 
	for(int i = beginAddress, j = 0; j < 32; ++i , ++j){
		u32 sharedAddress = target * 32 + j ; // 當前要交換的physical memory address
		u32 curAddress = frame_num * 32 + j ; // 想要交換到physical memory address 的page
		 
		storage[i] = buffer[sharedAddress] ;			//swap out 
		buffer[sharedAddress] = storage[curAddress];	//swap in 
	}
	int tmpTime = inTime++ ; 
	pt[target] = ((tmpTime) << 13 ) | ( frame_num << 1 ) | 1 ;
	return target * 32 + offset ;
}

__device__ void init_pageTable( int pt_entries ){
	for(int i = 0; i < pt_entries; ++i ){
		pt[i] = 0  ; 
	}

	for(int i = 0; i < STORAGE_SIZING; ++i){
		storage[i] = 0 ;
	}
}

int load_binaryFile( const char *DATAFILE, uchar *input, int STORAGE_SIZE ){
	int size = 0 ; 
	uchar in ; 
	FILE *R = fopen( DATAFILE, "rb" ) ; 
	
	while( fread( &in, sizeof( uchar ), 1, R ) )	
		input[size++] = in ; 
	fclose( R )  ;
	
	return size ;  
}

void write_binaryFile( const char *OUTFILE, uchar *results, int input_size ){
	FILE *W = fopen( OUTFILE, "wb" ) ; 
	for(int i = 0; i < input_size; ++i ){
		fwrite( &results[i], sizeof( uchar ), 1, W ) ;
	}
	fclose( W ) ; 
}
